#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void cuda_hello(){
    printf("Hello World from GPU!\n");
}

void showCUDAStats( void )
{
	// code from https://devblogs.nvidia.com/how-query-device-properties-and-handle-errors-cuda-cc/
	int nDevices;

	hipGetDeviceCount(&nDevices);
	for (int i = 0; i < nDevices; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		printf("Device Number: %d\n", i);
		printf("  Device name: %s\n", prop.name);
		printf("  Memory Clock Rate (KHz): %d\n",
		       prop.memoryClockRate);
		printf("  Memory Bus Width (bits): %d\n",
		       prop.memoryBusWidth);
		printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
		       2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
	}

	cuda_hello<<<1,1>>>();
	fflush(stdout);
}
